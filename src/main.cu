#include <stdlib.h>
#include <stdio.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#define index(i,j,ld) (((j)*(ld))+(i))

__global__ void copySubmatrix(float *gpu_X, float *sub_X, int *gpu_indices, int length_ind, int n, int p)
  
{
  int k = threadIdx.x + blockDim.x*blockIdx.x;
  if(k < n*length_ind){
    int j = (k - 1) / n;
    int i = k - n * j;    
    sub_X[j * n + i] = gpu_X[gpu_indices[j] * n + i];
  }
}


__global__ void copySubBeta(float *gpu_beta, float *gpu_Abeta, int *gpu_indices, int length_ind)
  
{
  int k = threadIdx.x + blockDim.x*blockIdx.x;
  if(k < length_ind){    
    gpu_Abeta[k] = gpu_beta[gpu_indices[k]];
  }
}

__global__ void copyunSubBeta(float *gpu_beta, float *gpu_Abeta, int *gpu_indices, int length_ind)
  
{
  int k = threadIdx.x + blockDim.x*blockIdx.x;
  if(k < length_ind){    
    gpu_beta[gpu_indices[k]] = gpu_Abeta[k];
  }
}

__global__ void checkKKT(float *gpu_grad, int *gpu_isActive, float lambda, int p)

{
  int k = threadIdx.x + blockDim.x*blockIdx.x;
  if(k < p){
    if((gpu_grad[k] < -lambda) || (gpu_grad[k] > lambda)){
      gpu_isActive[k] = 1;
    }
  }
}

//Extract ind-th element of gpu_vector
__global__ void getKernel (float *gpu_vector, int ind, float *gpu_val)

{
  gpu_val[0] = gpu_vector[ind];
}

__global__ void softKernel(float *gpu_beta, float lambda, int p)
  
{
  int k = threadIdx.x + blockDim.x*blockIdx.x;
  if(k < p){    
    if((gpu_beta[k] > -lambda) && (gpu_beta[k] < lambda)){
      gpu_beta[k] = 0;
    }
    else if(gpu_beta[k] > lambda){
      gpu_beta[k] = gpu_beta[k] - lambda;
    }
    else if(gpu_beta[k] < -lambda){
      gpu_beta[k] = gpu_beta[k] + lambda;
    }
  }
}


  
extern "C"{
 
  //copies part of gpu_X into sub_X
  void subMatrix(float *gpu_X, float *sub_X, int *gpu_indices, int length_ind, int n, int p){
    int block_size = 256;
    int n_blocks = n*length_ind/block_size + ((n*length_ind)%block_size == 0 ? 0:1);
    
    copySubmatrix <<< block_size, n_blocks >>> (gpu_X, sub_X, gpu_indices, length_ind, n, p);
  }
  
  void subBeta(float *gpu_beta, float *gpu_Abeta, int *gpu_indices, int length_ind){
    int block_size = 256;
    int n_blocks = length_ind/block_size + ((length_ind)%block_size == 0 ? 0:1);
    
    copySubBeta <<< block_size, n_blocks >>> (gpu_beta, gpu_Abeta, gpu_indices, length_ind);
  }

  void unsubBeta(float *gpu_beta, float *gpu_Abeta, int *gpu_indices, int length_ind){
    int block_size = 256;
    int n_blocks = length_ind/block_size + ((length_ind)%block_size == 0 ? 0:1);
    
    copyunSubBeta <<< block_size, n_blocks >>> (gpu_beta, gpu_Abeta, gpu_indices, length_ind);
  }

  void softThreshold(float *gpu_beta, float lambda, float step, int p){
    int block_size = 256;
    int n_blocks = p/block_size + ((p)%block_size == 0 ? 0:1);
    
    softKernel <<< block_size, n_blocks >>> (gpu_beta, lambda*step, p);
  }

  //transfers gpu_vector[ind] into returnPtr
  void getIndVal(float *gpu_vector, int ind, float *returnPtr){
    int block_size = 1;
    int n_blocks = 1;
    float *gpu_val;
    hipMalloc((void**) &gpu_val, sizeof(float));
    getKernel <<< block_size, n_blocks >>> (gpu_vector, ind, gpu_val);
    hipMemcpy(returnPtr, gpu_val, sizeof(float), hipMemcpyDeviceToHost);
    hipFree(gpu_val);
  }


  void checkStep(float *gpu_X, float *gpu_resid, float *gpu_grad, int* gpu_indices, int* indices, float lambda, int *cont, int *gpu_isActive, int *isActive, int *numActive, int *gpu_numActive, int *n, int *p){
    int i;
    int counter = 0;
    cont[0] = 0;
    int oldNumActive = numActive[0];

    /* Calculating new grad */
    cublasSgemv('t', n[0], p[0], 1, gpu_X, n[0], gpu_resid, 1, 0, gpu_grad, 1);

    /* Checking if KKT holds */

    int block_size = 256;
    int n_blocks = p[0]/block_size + ((p[0])%block_size == 0 ? 0:1);
    
    checkKKT <<< block_size, n_blocks >>> (gpu_grad, gpu_isActive, lambda, p[0]);
 
    numActive[0] = 0;

    hipMemcpy(isActive, gpu_isActive, sizeof(int)*p[0], hipMemcpyDeviceToHost);

    for(i=0; i<p[0];i++){
      if(isActive[i] != 0){
	    indices[counter] = i;
	    counter++;
      }
    }
    numActive[0] = counter;

    if(numActive[0] > oldNumActive){
      cont[0] = 1;
    }
    hipMemcpy(gpu_numActive, numActive, sizeof(int), hipMemcpyHostToDevice);


    hipMemcpy(gpu_indices, indices, sizeof(int)*p[0],hipMemcpyHostToDevice);

  }

  void gradStep(float *gpu_X, float *gpu_y, float *gpu_resid, float *gpu_fit, float *gpu_beta, float *gpu_oldBeta, float *gpu_grad, float *gpu_diff, float lambda, float *thresh, int *maxIt, float *step_size, float *beta, int *n, int *p, float *diff, float *step){
 
    float oldLL = 0;
    float newLL = 0;
    float dot_val = 0;
    int max_move_ind = 0;
    float *max_move;
    max_move = (float*)malloc(sizeof(float));
    max_move[0] = 0;

    /* Copying beta to oldBeta for backtracking */
    cublasScopy(p[0], gpu_beta, 1, gpu_oldBeta, 1);
    
    /* Calculating the new fit */
    cublasSgemv('n', n[0], p[0], 1, gpu_X, n[0], gpu_beta, 1, 0, gpu_fit, 1);

    /* Calculating new residuals */
    cublasScopy(n[0], gpu_y, 1, gpu_resid, 1);  // Copying y to resid
    cublasSaxpy(n[0], -1, gpu_fit, 1, gpu_resid, 1);  // Subtracting fit from y (which is stored in resid)

     /* Calculating oldLL based on resid */

    oldLL = cublasSnrm2(n[0], gpu_resid, 1);
  
    /* Calculating new grad */
    cublasSgemv('t', n[0], p[0], 1, gpu_X, n[0], gpu_resid, 1, 0, gpu_grad, 1);

     /* Step beta in the proper direction */
    cublasSaxpy(p[0], step_size[0], gpu_grad, 1, gpu_beta, 1);

    /* Soft-threshholding beta by lambda */
    
    softThreshold(gpu_beta, lambda, step_size[0], p[0]);
    
    /* Step size optimization */
    // Calculating RHS
    /* Calculating difference between beta and oldBeta */
    
    cublasScopy(p[0], gpu_beta, 1, gpu_diff, 1);
    cublasSaxpy(p[0], -1, gpu_oldBeta, 1, gpu_diff, 1);
    
    /* calculating the dot product between grad and diff */
    
    dot_val = cublasSdot(p[0], gpu_diff, 1, gpu_grad, 1);
    
    /* Calculating length of move */
    
    *step = cublasSnrm2(p[0], gpu_diff, 1);
    max_move_ind = cublasIsamax(p[0], gpu_diff, 1); /// Problem???
    
    /* Terrible way to do this! Don't need to copy the whole vector! */
    
    getIndVal(gpu_diff, (max_move_ind-1), max_move);
    max_move[0] = max_move[0] * max_move[0];

    //   cublasGetVector(p[0], sizeof(float), gpu_diff, 1, diff, 1);
    //max_move = diff[max_move_ind-1]*diff[max_move_ind-1];
    
    // Calculating LHS
    
    cublasSgemv('n', n[0], p[0], 1, gpu_X, n[0], gpu_beta, 1, 0, gpu_fit,1);
    cublasScopy(n[0], gpu_y, 1, gpu_resid, 1);
    cublasSaxpy(n[0], -1, gpu_fit, 1, gpu_resid, 1);
    newLL = cublasSnrm2(n[0], gpu_resid, 1);
    
    if(newLL*newLL/2 > oldLL*oldLL/2 - dot_val + step[0]*step[0]/(2*step_size[0])){
      cublasScopy(p[0], gpu_oldBeta, 1, gpu_beta, 1);
      step_size[0] = step_size[0] * 0.8;
      step[0] = 100000;
    }
    free(max_move);
    }



  
  void singleSol(float *gpu_X, float *gpu_y, float *gpu_resid, float *gpu_fit, float *gpu_beta, float *gpu_oldBeta, float *gpu_grad, float *gpu_diff, float lambda, float *thresh, int *maxIt, float *step_size_set, float *beta, int *n, int *p, float *diff, int* gpu_isActive, int* isActive, int* numActive, int* gpu_numActive, int* gpu_indices, int *indices,float* gpu_AX, float* gpu_Abeta, float* gpu_AoldBeta, float* gpu_Agrad, float* gpu_Adiff, float* Abeta, float* Adiff){
  
  int count = 0;
  int cont = 1;
  int inner_cont = 1; // inner loop variable (for active set)
  float step = 0;
  float init_step = step_size_set[0];

  int act_p = numActive[0];

  checkStep(gpu_X, gpu_resid, gpu_grad, gpu_indices, indices, lambda, &cont, gpu_isActive, isActive, numActive, gpu_numActive, n, p);
 
  while(cont == 1){
    inner_cont = 1;
    /* Defining all the new active variables */
  
    subBeta(gpu_beta, gpu_Abeta, gpu_indices, numActive[0]);
 
    subMatrix(gpu_X, gpu_AX, gpu_indices, numActive[0], n[0], p[0]);    

    while(inner_cont == 1){    
      
      act_p = numActive[0];
      gradStep(gpu_AX, gpu_y, gpu_resid, gpu_fit, gpu_Abeta, gpu_AoldBeta, gpu_Agrad, gpu_Adiff, lambda, thresh, maxIt, step_size_set, Abeta, n, &act_p, Adiff, &step);


      /* Checking if stop criteria are satisfied */
      count++;
      if(count > maxIt[0]){
	inner_cont = 0;
      }
      if(step < thresh[0]){ // Switch to max_move
	inner_cont = 0;
      }
    }
  

    unsubBeta(gpu_beta, gpu_Abeta, gpu_indices, numActive[0]);


    checkStep(gpu_X, gpu_resid, gpu_grad, gpu_indices, indices, lambda, &cont, gpu_isActive, isActive, numActive, gpu_numActive, n, p);
    
  }
  step_size_set[0] = init_step;
  //Rprintf("%u ", count);
  }
  
 



void activePathSol(float* X, float* y, int* n, int* p, int* maxIt, float* thresh, float* step_size, float* lambda, float* beta, int* num_lambda){ 

  int number_of_devices;
  hipGetDeviceCount(&number_of_devices);
  //Rprintf("%u ", number_of_devices);
  hipSetDevice(0);

  int i,j;
  hipblasStatus_t status;

  cublasInit();

  /* ALLOCATING HOST MEMORY */
 
  float *grad = (float*)malloc(p[0]*sizeof(float));
  float *oldBeta = (float*)malloc(p[0]*sizeof(float));
  float *workingBeta = (float*)malloc(p[0]*sizeof(float));
  float *fit = (float*)malloc(n[0]*sizeof(float));
  float *resid = (float*)malloc(n[0]*sizeof(float));
  float *diff = (float*)malloc(p[0]*sizeof(float));
  int *isActive = (int*)malloc(p[0]*sizeof(int));
  int *numActive = (int*)malloc(sizeof(int));
  int *indices = (int*)malloc(p[0]*sizeof(float)); // Ever active index

  /* INITIALIZING ARRAY VALUES */
  
  for (i=0;i<n[0];i++){
    resid[i] = y[i];
    fit[i] = 0;
  }
  for (i=0;i<p[0];i++){
    grad[i] = 0;
    oldBeta[i] = 0;
    isActive[i] = 0;
    indices[i] = -1;
  }
  numActive[0] = 0;

  /* INITIALIZING POINTERS FOR THE GPU VERSIONS OF VARIABLES */

  float* gpu_X; float* gpu_y; float* gpu_workingBeta; float* gpu_oldBeta; float* gpu_fit; float* gpu_resid; float* gpu_grad; float* gpu_diff; int* gpu_isActive;  int* gpu_numActive; int* gpu_indices;

  /* ALLOCATING MEMORY ON THE GPU */

  status=cublasAlloc(n[0]*p[0],sizeof(float),(void**)&gpu_X);
  status=cublasAlloc(n[0],sizeof(float),(void**)&gpu_y);
  status=cublasAlloc(n[0],sizeof(float),(void**)&gpu_resid);
  status=cublasAlloc(n[0],sizeof(float),(void**)&gpu_fit);
  status=cublasAlloc(p[0],sizeof(float),(void**)&gpu_workingBeta);
  status=cublasAlloc(p[0],sizeof(float),(void**)&gpu_oldBeta);
  status=cublasAlloc(p[0],sizeof(float),(void**)&gpu_grad);
  status=cublasAlloc(p[0],sizeof(float),(void**)&gpu_diff);
  status=cublasAlloc(p[0],sizeof(int),(void**)&gpu_isActive);
  status=cublasAlloc(p[0], sizeof(int),(void**)&gpu_indices);
  hipMalloc((void**) &gpu_numActive, sizeof(int));

  /* Defining submatrix/activeset stuff */

  float *Abeta = (float*)malloc(p[0]*sizeof(float));
  float *Adiff = (float*)malloc(p[0]*sizeof(float));
 
  float *gpu_AX; float *gpu_Abeta; float *gpu_AoldBeta; float *gpu_Agrad;  float *gpu_Adiff;
  cublasAlloc(n[0]*p[0],sizeof(float),(void**)&gpu_AX);
  cublasAlloc(p[0], sizeof(int),(void**)&gpu_Abeta);
  cublasAlloc(p[0], sizeof(int),(void**)&gpu_AoldBeta);
  cublasAlloc(p[0], sizeof(int),(void**)&gpu_Agrad);
  cublasAlloc(p[0], sizeof(int),(void**)&gpu_Adiff);

  /* MOVING THE MATRICES OVER TO GPU MEMORY */

  status=hipblasSetMatrix(n[0],p[0],sizeof(float),X,n[0],gpu_X,n[0]);
  status=hipblasSetVector(n[0],sizeof(float),y,1,gpu_y,1);
  status=hipblasSetVector(n[0],sizeof(float),resid,1,gpu_resid,1);
  status=hipblasSetVector(n[0],sizeof(float),fit,1,gpu_fit,1);
  status=hipblasSetVector(p[0],sizeof(float),oldBeta,1,gpu_workingBeta,1);
  status=hipblasSetVector(p[0],sizeof(float),oldBeta,1,gpu_oldBeta,1);
  status=hipblasSetVector(p[0],sizeof(float),grad,1,gpu_grad,1);
  status=hipblasSetVector(p[0],sizeof(int),isActive,1,gpu_isActive,1);
  status=hipblasSetVector(p[0],sizeof(int),indices,1,gpu_indices,1);
  hipMemcpy(gpu_numActive, numActive, sizeof(int), hipMemcpyHostToDevice);

  /* RUNNING A LOOP TO SOVLE FOR EACH LAMBDA */

  for(j=0; j < num_lambda[0]; j++){
 
    singleSol(gpu_X, gpu_y, gpu_resid, gpu_fit, gpu_workingBeta, gpu_oldBeta, gpu_grad, gpu_diff, lambda[j], thresh, maxIt, step_size, workingBeta, n, p, diff, gpu_isActive, isActive, numActive, gpu_numActive, gpu_indices, indices, gpu_AX, gpu_Abeta, gpu_AoldBeta, gpu_Agrad, gpu_Adiff, Abeta, Adiff);

    hipblasGetVector(p[0], sizeof(float), gpu_workingBeta, 1, workingBeta, 1);

    numActive[0] = 0;
    hipMemcpy(gpu_numActive, numActive, sizeof(int), hipMemcpyHostToDevice);

    /* END OF Shouldn't be necessary!!!*/

    
    /* STORING CURRENT BETA VALUE IN BETA */
    for(i=0; i < p[0]; i++){
      beta[j*p[0]+i] = workingBeta[i];
    }
  }
  
  /* FREEING UP MEMORY */

  free ( grad ); free( fit ); free( resid ); free( oldBeta ); free( workingBeta ); free( diff ); free ( numActive ); free( Abeta ); free( Adiff ); free( indices ); free( isActive );
  status = cublasFree(gpu_X);
  status = cublasFree(gpu_y);
  status = cublasFree(gpu_grad);
  status = cublasFree(gpu_workingBeta);
  status = cublasFree(gpu_oldBeta);
  status = cublasFree(gpu_resid);
  status = cublasFree(gpu_fit);
  status = cublasFree(gpu_diff);
  status = cublasFree(gpu_isActive);
  status = cublasFree(gpu_indices);
  hipFree(gpu_numActive);
  cublasFree(gpu_AX);
  cublasFree(gpu_Agrad);
  cublasFree(gpu_Abeta);
  cublasFree(gpu_AoldBeta);
  cublasFree(gpu_Adiff);
  
 /* Shutdown */
  status = cublasShutdown();
}
}
